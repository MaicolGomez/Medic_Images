#include "hip/hip_runtime.h"
#include<bits/stdc++.h>
using namespace std;
#define pi (2.0*acos(0.0))
#define eps 1e-6
#define ll long long
#define inf (1<<29)
#define vi vector<int>
#define vll vector<ll>
#define sc(x) scanf("%d",&x)
#define scl(x) scanf("%lld",&x)
#define all(v) v.begin() , v.end()
#define me(a,val) memset( a , val ,sizeof(a) )
#define pb(x) push_back(x)
#define pii pair<int,int> 
#define mp(a,b) make_pair(a,b)
#define Q(x) (x) * (x)
#define L(x) ((x<<1) + 1)
#define R(x) ((x<<1) + 2)
#define M(x,y) ((x+y)>>1)
#define fi first
#define se second
#define MOD 1000000007
#define ios ios::sync_with_stdio(0)
#define N 100000


__global__ void f(double *A,double *B,double *ans,int n,int d,int need){
	
	int i = threadIdx.x;
	while( i < n ){
		
		
		__syncthreads();
		i += blockDim.x * gridDim.x;
	}
	
}

int main(){
	int n , d , need;
	scanf("%d%d%d",&n,&d,&need);
	double C[d];
	for(int j = 0 ; j < d ; j++)
		scanf("%lf",&C[j]);
		
	double A[d] , *B;
	hipMalloc( (void **)&B , sizeof(double) * d );
	
	for(int i = 0 ; i < n ; i++){
		double dis = 0;
		for(int j = 0 ; j < d ; j++)
			scanf("%lf",&A[j]);
		
		hipMemcpy( B , A , sizeof(double) * n * d , hipMemcpyHostToDevice );
		
	}
	
	f<<< 1 , 1000 >>>( B , D , ans , n , d , need );
	
	hipFree( B );
	hipFree( D );
	return 0;
}
