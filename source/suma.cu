
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;
#define pi (2.0*acos(0.0))
#define eps 1e-6
#define ll long long
#define inf (1<<29)
#define vi vector<int>
#define vll vector<ll>
#define sc(x) scanf("%d",&x)
#define scl(x) scanf("%lld",&x)
#define all(v) v.begin() , v.end()
#define me(a,val) memset( a , val ,sizeof(a) )
#define pb(x) push_back(x)
#define pii pair<int,int> 
#define mp(a,b) make_pair(a,b)
#define Q(x) (x) * (x)
#define L(x) ((x<<1) + 1)
#define R(x) ((x<<1) + 2)
#define M(x,y) ((x+y)>>1)
#define fi first
#define se second
#define MOD 1000000007
#define ios ios::sync_with_stdio(0)
#define N 1024
#define BL 32

#define NA 20

__global__ void suma(int *A, int *S){
	
	S[0] = S[0] + A[threadIdx.x];
	printf("A[t]: %d S[0]: %d\n",A[threadIdx.x], S[0]);
	
	__syncthreads();
}

int main(){

	int *a = new int[NA];	
	
	for(int i = 0; i < NA; i++) a[i] = 1;
	
	int *A;
	
	hipMalloc(&A, NA * sizeof(int));
	hipMemcpy(A, a, NA * sizeof(int), hipMemcpyHostToDevice);
	
	int *s = new int[1];
	s[0] = 0;
	int *S;
	
	hipMalloc(&S, sizeof(int));
	hipMemcpy(S, s, sizeof(int), hipMemcpyHostToDevice);
	
	
	suma<<<1,NA>>>(A, S);
	
	cout<<"llego"<<endl;
	
	hipMemcpy(s, S, sizeof(int), hipMemcpyDeviceToHost);
	
	
	
	cout<<s[0]<<endl;
	
	hipFree(A);
	hipFree(S);
	
	return 0;
}
