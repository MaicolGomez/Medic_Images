
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;
#define pi (2.0*acos(0.0))
#define eps 1e-6
#define ll long long
#define inf (1<<29)
#define vi vector<int>
#define vll vector<ll>
#define sc(x) scanf("%d",&x)
#define scl(x) scanf("%lld",&x)
#define all(v) v.begin() , v.end()
#define me(a,val) memset( a , val ,sizeof(a) )
#define pb(x) push_back(x)
#define pii pair<int,int> 
#define mp(a,b) make_pair(a,b)
#define Q(x) (x) * (x)
#define L(x) ((x<<1) + 1)
#define R(x) ((x<<1) + 2)
#define M(x,y) ((x+y)>>1)
#define fi first
#define se second
#define MOD 1000000007
#define ios ios::sync_with_stdio(0)
#define TRADITIONAL 1
#define N 1024

typedef struct MatrixStruct {
	double **data;
	int height, width;
} Matrix;

void nrerror(string s){
	printf("Numerical Recipes run-time error...\n");
	printf("%s\n",s.c_str());
	printf("...now exiting to system...\n");
	exit(1);
}

double *dvector(int nl, int nh){
	double *v;
	v = (double *) calloc((unsigned) (nh-nl+1), sizeof(double));
	if (!v) nrerror("allocation failure in dvector()");
	return v-nl;
}

float *vector(int nl, int nh){
	float *v;
	v = (float *) calloc((unsigned) (nh-nl+1), sizeof(float));
	if (!v) nrerror("allocation failure in dvector()");
	return v-nl;
}

void free_dvector(double *v, int nl, int nh){
	free((char*) (v+nl));
}

void free_vector(float *v, int nl, int nh){
	free((char*) (v+nl));
}

float **matrix(int nrl,int nrh,int ncl,int nch){
	float **m;
	m = (float **) malloc((unsigned) (nrh-nrl+1)*sizeof(float*));
	if (!m) nrerror("allocation failure 1 in matrix()");
	m -= nrl;
	for(int i = nrl ; i <= nrh ; i++){
		m[i]=(float *) malloc((unsigned) (nch-ncl+1)*sizeof(float));
		if (!m[i]) nrerror("allocation failure 2 in matrix()");
		m[i] -= ncl;
	}
	return m;
}

void free_matrix(float **m,int nrl,int nrh,int ncl,int nch){
	for(int i = nrh ; i >= nrl ; i--) free((char*) (m[i]+ncl));
	free((char*) (m+nrl));
}

double **dmatrix(int nrl, int nrh, int ncl, int nch){
	double **m;
	m = (double **) calloc((unsigned) (nrh-nrl+1), sizeof(double*));
	if (!m) nrerror("allocation failure 1 in dmatrix()");
	m -= nrl;
	for(int i = nrl ; i <= nrh ; i++){
		m[i] = (double *) calloc((unsigned) (nch-ncl+1), sizeof(double));
		if (!m[i]) nrerror("allocation failure 2 in dmatrix()");
		m[i] -= ncl;
	}
	return m;
}

void free_dmatrix(double **m, int nrl, int nrh, int ncl, int nch){
	for(int i = nrh ; i >= nrl ; i--) free((char*) (m[i]+ncl));
	free((char*) (m+nrl));
}

/*
double log2(double x){
	return log10(x) / log10( 2.0 );
}
*/

void CreateMatrix(Matrix **M, int hei, int wid){
	Matrix *tmp;

	tmp = (Matrix *) calloc(1, sizeof(Matrix)); 
	tmp->data = (double **) calloc(hei, sizeof(double *));
	
	if (!(tmp->data)) {
		nrerror("allocation failure in CreateMatrix()");
		exit(1);
	}

	for (int h = 0 ; h < hei ; h++) {
		tmp->data[h] = (double *) calloc(wid, sizeof(double));
		if (!(tmp->data[h])) {
			nrerror("allocation failure in CreateMatrix()");
			exit(1);
		}
	}

	tmp->height = hei;
	tmp->width = wid;
	*M = tmp;
}

void FreeMatrix(Matrix *M){
	int hei = M->height;
	for(int h = 0 ; h < hei ; h++){
	     free(M->data[h]);
	}
	free(M->data);
	free(M);
}


void four1(double *data, int nn, int isign){
	int n, mmax, m, j, istep, i;
	double wtemp, wr, wpr, wpi, wi, theta;
	double tempr, tempi;
	n = nn << 1;
	j = 1;
	for (i=1;i<n;i+=2) {
		if (j > i) {
			swap(data[j],data[i]);
			swap(data[j+1],data[i+1]);
		}
		m = n >> 1;
		while (m >= 2 && j > m) {
			j -= m;
			m >>= 1;
		}
		j += m;
	}
	mmax = 2;
	while (n > mmax) {
		istep = 2*mmax;
		theta = 6.28318530717959/(isign*mmax);
		wtemp = sin(0.5*theta);
		wpr = -2.0*wtemp*wtemp;
		wpi = sin(theta);
		wr = 1.0;
		wi = 0.0;
		for (m=1;m<mmax;m+=2) {
			for (i=m;i<=n;i+=istep) {
				j = i+mmax;
				tempr = wr*data[j]-wi*data[j+1];
				tempi = wr*data[j+1]+wi*data[j];
				data[j] = data[i]-tempr;
				data[j+1] = data[i+1]-tempi;
				data[i] += tempr;
				data[i+1] += tempi;
			}
			wr = (wtemp=wr)*wpr-wi*wpi+wr;
			wi = wi*wpr+wtemp*wpi+wi;
		}
		mmax = istep;
	}
}

void four2(double **fftr, double **ffti, double **rdata, double **idata, int rs, int cs, int isign){
/************************************************************ 

   2-D fourier transform of data with real part stored in
   "rdata" and imaginary part in "idata" with size "rs" x
   "cs". The result is in "fftr" and "ffti". The isign is
   "isign" =  1 forward, and "isign" = -1 inverse 

*************************************************************/
        double **T, *tmp1, *tmp2;
        int i, j;

        tmp1 = dvector(1,2*cs);
        tmp2 = dvector(1,2*rs);
        T = dmatrix(1,2*rs,1,cs);

        for (i=1;i<=rs;i++) {
            for (j=1;j<=cs;j++) {
                tmp1[j*2-1] = rdata[i][j];
                tmp1[j*2] = idata[i][j];
            }
            four1(tmp1, cs, isign);
            for (j=1;j<=cs;j++) {
                T[i*2-1][j] = tmp1[j*2-1];
                T[i*2][j] = tmp1[j*2];
            }
        }

        for (i=1;i<=cs;i++) {
            for (j=1;j<=rs;j++) {
                tmp2[j*2-1] = T[j*2-1][i];
                tmp2[j*2] = T[j*2][i];
            }
            four1(tmp2,rs,isign);
            for (j=1;j<=rs;j++) {
                fftr[j][i] = tmp2[j*2-1];
                ffti[j][i] = tmp2[j*2];
            }
        }
        free_dvector(tmp1, 1, 2*cs);
        free_dvector(tmp2, 1, 2*rs);
        free_dmatrix(T, 1, 2*rs, 1, cs); 
}

void Mat_FFT2(Matrix *Output_real, Matrix *Output_imag, Matrix *Input_real, Matrix *Input_imag){
	int xs, ys, i, j;
	double **R, **I, **Fr, **Fi;

	xs = Input_real->height;
	ys = Input_real->width;

    R  = dmatrix(1,xs,1,ys);
    I  = dmatrix(1,xs,1,ys);
    Fr = dmatrix(1,xs,1,ys);
    Fi = dmatrix(1,xs,1,ys);
		
    for (i=1;i<=Input_real->height;i++) 
        for (j=1;j<=Input_real->width;j++) {
            R[i][j] = Input_real->data[i-1][j-1];
            I[i][j] = Input_imag->data[i-1][j-1];
        }

    four2(Fr, Fi, R, I, xs, ys, 1);         /* 2-D FFT */

    for (i=1;i<=Input_real->height;i++) 
        for (j=1;j<=Input_real->width;j++) {
            Output_real->data[i-1][j-1] = Fr[i][j];
            Output_imag->data[i-1][j-1] = Fi[i][j];
        }

    free_dmatrix(R,1,xs,1,ys);
    free_dmatrix(I,1,xs,1,ys);   
    free_dmatrix(Fr,1,xs,1,ys);
    free_dmatrix(Fi,1,xs,1,ys);   
}

void Mat_IFFT2(Matrix *Output_real, Matrix *Output_imag, Matrix *Input_real, Matrix *Input_imag){
	int xs, ys, i, j;
	double **R, **I, **Fr, **Fi, NN;

	xs = Input_real->height;
	ys = Input_real->width;

    R  = dmatrix(1,xs,1,ys);
    I  = dmatrix(1,xs,1,ys);
    Fr = dmatrix(1,xs,1,ys);
    Fi = dmatrix(1,xs,1,ys);

    for (i=1;i<=Input_real->height;i++) 
        for (j=1;j<=Input_real->width;j++) {
            R[i][j] = Input_real->data[i-1][j-1];
            I[i][j] = Input_imag->data[i-1][j-1];
        }

    four2(Fr, Fi, R, I, xs, ys, -1);         /* 2-D IFFT */

	NN = (double) (xs*ys);

    for (i=1;i<=Input_real->height;i++) 
        for (j=1;j<=Input_real->width;j++) {
            Output_real->data[i-1][j-1] = Fr[i][j]/NN;
            Output_imag->data[i-1][j-1] = Fi[i][j]/NN;
        }

    free_dmatrix(R,1,xs,1,ys);
    free_dmatrix(I,1,xs,1,ys);   
    free_dmatrix(Fr,1,xs,1,ys);
    free_dmatrix(Fi,1,xs,1,ys);   
}

void Mat_Copy(Matrix *A, Matrix *B, int h_target, int w_target, int h_begin, int w_begin, int h_end, int w_end){
	int i, j, h, w, h_done, w_done;
	if ((h_target >= 0)&&(h_target < A->height)&&(w_target >= 0)&&(w_target < A->width)) {
		if ((h_begin >= 0)&&(h_begin < B->height)&&(w_begin >= 0)&&(w_begin < B->width)) {
			h = h_end-h_begin+1;
			w = w_end-w_begin+1;
			if ((h >= 1)&&(w >= 1)) {
				h_done = h_target+h-1;
				w_done = w_target+w-1;
				if ((h_done < A->height)&&(w_done < A->width)) {
					for (i=0;i<h;i++) {
						for (j=0;j<w;j++) {
							A->data[i+h_target][j+w_target] = B->data[i+h_begin][j+w_begin];
						}
					}
				}
			}
		}
	}
	else {
		printf("matrix dimension error!\n");
		exit(1);
	}
}


void Mat_Product(Matrix *A, Matrix *B, Matrix *C){
	for(int h = 0 ; h < A->height ; h++)
		for(int w = 0 ; w < A->width ; w++)
			A->data[h][w] = B->data[h][w]*C->data[h][w];
}

void Mat_Sum(Matrix *A, Matrix *B, Matrix *C){
	for(int h = 0 ; h < A-> height ; h++)
		for(int w = 0 ; w < A->width ; w++)
			A->data[h][w] = B->data[h][w]+C->data[h][w];
}

void Mat_Substract(Matrix *A, Matrix *B, Matrix *C){
	for(int h = 0 ; h < A->height ; h++)
		for(int w = 0 ; w < A->width ; w++)
			A->data[h][w] = B->data[h][w]-C->data[h][w];
}

void Gabor(Matrix *Gr, Matrix *Gi, int s, int n, double Ul, double Uh, int scale, int orientation, int flag);

void GaborFilteredImg(Matrix *FilteredImg_real, Matrix *FilteredImg_imag, Matrix *img, int side, double Ul, double Uh, int scale, int orientation, int flag){
	int h, w, xs, ys, border, r1, r2, r3, r4, hei, wid, s, n, base;
	Matrix *IMG, *IMG_imag, *Gr, *Gi, *Tmp_1, *Tmp_2, *F_1, *F_2, *G_real, *G_imag, *F_real, *F_imag,*F;
	double m, v;
	
	base = scale*orientation;
	double *features; //exact memory
	features = (double *)malloc( 2 * scale * orientation * sizeof(double) );

	border = side;
	hei = img->height;
	wid = img->width;

	/* FFT2 */
	xs = (int) pow(2.0, ceil(log2((double)(img->height+2.0*border))));
	ys = (int) pow(2.0, ceil(log2((double)(img->width+2.0*border))));

	CreateMatrix(&IMG, xs, ys);

	r1 = img->width+border;
	r2 = img->width+2*border;
	for (h=0;h<border;h++) {
		for (w=0;w<border;w++)
			IMG->data[h][w] = img->data[border-1-h][border-1-w];
		for (w=border;w<r1;w++)
			IMG->data[h][w] = img->data[border-1-h][w-border];
		for (w=r1;w<r2;w++)
			IMG->data[h][w] = img->data[border-1-h][2*img->width-w+border-1];
	}

	r1 = img->height+border;
	r2 = img->width+border;
	r3 = img->width+2*border;
	for (h=border;h<r1;h++) {
		for (w=0;w<border;w++)
			IMG->data[h][w] = img->data[h-border][border-1-w];
		for (w=border;w<r2;w++)
			IMG->data[h][w] = img->data[h-border][w-border];
		for (w=r2;w<r3;w++)
			IMG->data[h][w] = img->data[h-border][2*img->width-w+border-1];
	}

	r1 = img->height+border;
	r2 = img->height+2*border;
	r3 = img->width+border;
	r4 = img->width+2*border;
	for (h=r1;h<r2;h++) {
		for (w=0;w<border;w++)
			IMG->data[h][w] = img->data[2*img->height-h+border-1][border-1-w];
		for (w=border;w<r3;w++)
			IMG->data[h][w] = img->data[2*img->height-h+border-1][w-border];
		for (w=r3;w<r4;w++)
			IMG->data[h][w] = img->data[2*img->height-h+border-1][2*img->width-w+border-1];
	}

	CreateMatrix(&F_real, xs, ys);
	CreateMatrix(&F_imag, xs, ys);
	CreateMatrix(&IMG_imag, xs, ys);

	Mat_FFT2(F_real, F_imag, IMG, IMG_imag);

	/* ----------- compute the Gabor filtered output ------------- */

	CreateMatrix(&Gr, 2*side+1, 2*side+1);
	CreateMatrix(&Gi, 2*side+1, 2*side+1);
	CreateMatrix(&Tmp_1, xs, ys);
	CreateMatrix(&Tmp_2, xs, ys);
	CreateMatrix(&F_1, xs, ys);
	CreateMatrix(&F_2, xs, ys);
	CreateMatrix(&G_real, xs, ys);
	CreateMatrix(&G_imag, xs, ys);
    CreateMatrix(&F, hei, wid);
    
	for (s=0;s<scale;s++){
		for (n=0;n<orientation;n++) {
			Gabor(Gr, Gi, s+1, n+1, Ul, Uh, scale, orientation, flag);//CUDA- 2 normales y logn
			Mat_Copy(F_1, Gr, 0, 0, 0, 0, 2*side, 2*side);//CUDA
			Mat_Copy(F_2, Gi, 0, 0, 0, 0, 2*side, 2*side);//CUDA
			Mat_FFT2(G_real, G_imag, F_1, F_2);//CUDA-no definido

			Mat_Product(Tmp_1, G_real, F_real);//CUDA
			Mat_Product(Tmp_2, G_imag, F_imag);//CUDA
			Mat_Substract(IMG, Tmp_1, Tmp_2);//CUDA

			Mat_Product(Tmp_1, G_real, F_imag);//CUDA
			Mat_Product(Tmp_2, G_imag, F_real);//CUDA
			Mat_Sum(IMG_imag, Tmp_1, Tmp_2);//CUDA

			Mat_IFFT2(Tmp_1, Tmp_2, IMG, IMG_imag);//CUDA-no definido
			
			//CUDA - logn
			m = 0;
			for (h=0;h<hei;h++)
				for (w=0;w<wid;w++) {
					F->data[h][w] = sqrt(pow(IMG->data[h][w], 2.0)+pow(IMG_imag->data[h][w], 2.0));
					m += F->data[h][w];
				}

			m /= (double) (hei*wid);
			features[s*orientation+n] = (float) m;
			
			//CUDA - logn
			v = 0;
			for (h=0;h<hei;h++)
				for (w=0;w<wid;w++)
					v += (F->data[h][w]-m)*(F->data[h][w]-m);

			v /= (double) (hei*wid);
			features[base+s*orientation+n] = (float) sqrt(v);
            
			Mat_Copy(FilteredImg_real, Tmp_1, s*hei, n*wid, 2*side, 2*side, hei+2*side-1, wid+2*side-1);//CUDA
			Mat_Copy(FilteredImg_imag, Tmp_2, s*hei, n*wid, 2*side, 2*side, hei+2*side-1, wid+2*side-1);//CUDA
		}
	}
	
    for(int i = 0 ; i < (2 * scale * orientation) ; i++)
        printf("%.8lf ",features[i]);
    printf("\n");
    cout << 2 * scale  * orientation << endl;
    
	FreeMatrix(Gr);
	FreeMatrix(Gi);
	FreeMatrix(Tmp_1);
	FreeMatrix(Tmp_2);
	FreeMatrix(F_1);
	FreeMatrix(F_2);
	FreeMatrix(G_real);
	FreeMatrix(G_imag);
	FreeMatrix(F_real);
	FreeMatrix(F_imag);
	FreeMatrix(IMG);
	FreeMatrix(IMG_imag);
}

/* ------------------------------------------------------------------------------------------------------
The Gabor function generates a Gabor filter with the selected index 's' and 'n' (scale and orientation, 
respectively) from a Gabor filter bank. This filter bank is designed by giving the range of spatial 
frequency (Uh and Ul) and the total number of scales and orientations used to partition the spectrum. 

The returned filter is stored in 'Gr' (real part) and 'Gi' (imaginary part).
--------------------------------------------------------------------------------------------------------*/
void Gabor(Matrix *Gr, Matrix *Gi, int s, int n, double Ul, double Uh, int scale, int orientation, int flag){
	double base, a, u0, z, Uvar, Vvar, Xvar, Yvar, X, Y, G, t1, t2, m;
	int x, y, side;

	base = Uh/Ul;
	a = pow(base, 1.0/(double)(scale-1));

	u0 = Uh/pow(a, (double) scale-s);

	Uvar = (a-1.0)*u0/((a+1.0)*sqrt(2.0*log(2.0)));

	z = -2.0*log(2.0)*(Uvar*Uvar)/u0;
	Vvar = tan(pi/(2*orientation))*(u0+z)/sqrt(2.0*log(2.0)-z*z/(Uvar*Uvar));

        Xvar = 1.0/(2.0*pi*Uvar);
        Yvar = 1.0/(2.0*pi*Vvar);

	t1 = cos(pi/orientation*(n-1.0));
	t2 = sin(pi/orientation*(n-1.0));

	side = (int) (Gr->height-1)/2;

	//CUDA
	for (x=0;x<2*side+1;x++) {
		for (y=0;y<2*side+1;y++) {
			X = (double) (x-side)*t1+ (double) (y-side)*t2;
			Y = (double) -(x-side)*t2+ (double) (y-side)*t1;
			G = 1.0/(2.0*pi*Xvar*Yvar)*pow(a, (double) scale-s)*exp(-0.5*((X*X)/(Xvar*Xvar)+(Y*Y)/(Yvar*Yvar)));
			Gr->data[x][y] = G*cos(2.0*pi*u0*X);
			Gi->data[x][y] = G*sin(2.0*pi*u0*X);
		}
	}

	/* if flag = 1, then remove the DC from the filter */
	

	if (flag == 1) {
	
		//CUDA - logn
		m = 0;
		for (x=0;x<2*side+1;x++)
			for (y=0;y<2*side+1;y++)
				m += Gr->data[x][y];

		m /= pow((double) 2.0*side+1, 2.0);
		
		
		//CUDA
		for (x=0;x<2*side+1;x++)
			for (y=0;y<2*side+1;y++)
				Gr->data[x][y] -= m;
	}	
}

int main(int argc, char **argv){
	int hei, wid, side, scale, orientation, flag;//, s, n;
	//Matrix *Gabor_r, *Gabor_i, *Gr, *Gi, *img, *F_r, *F_i;
	Matrix *img , *F_r , *F_i;
	FILE *fp;
	unsigned char *tmp; 
	//float *output;
	double Ul, Uh;
	/* --------------------------- Example --------------------------------
		scale = 3, 
		orientation = 4, 
		Uh (highest spatial frequency) = 0.4, 
		Ul (lowest spatial frequency) = 0.1,
		flag (removing the DC term) = 0 (False),
		side (filter dimension = (2*side+1)*(2*side+1)) = 60
	----------------------------------------------------------------------- */
	scale = 3;
	orientation = 4;
	Ul = 0.1;
	Uh = 0.4;
	flag = 0;
	side = 60;

	if (argc != 4) {
		printf("usage: %s <image_name> <height> <width>\n",argv[0]);
		exit(0);
	}
	
	hei = atoi(argv[2]);
	wid = atoi(argv[3]);

	tmp = (unsigned char *) calloc(hei*wid, sizeof(unsigned char));

	if ((fp = fopen(argv[1],"r")) == NULL) {
		printf("%s can not be open!\n", argv[1]);
		exit(0);
	}
	fread(tmp, sizeof(unsigned char), hei*wid, fp);
	fclose(fp);

	
	CreateMatrix(&img, hei, wid);
	for(int i = 0 ; i < hei ; i++)
		for(int j = 0 ; j < wid ; j++)
			img->data[i][j] = (double) (tmp[i*wid+j]);
			
	free(tmp);
		
	CreateMatrix(&F_r, hei*scale, wid*orientation);
	CreateMatrix(&F_i, hei*scale, wid*orientation);

	GaborFilteredImg(F_r, F_i, img, side, Ul, Uh, scale, orientation, flag);

	FreeMatrix(F_r);
	FreeMatrix(F_i);
	return 0;
}
