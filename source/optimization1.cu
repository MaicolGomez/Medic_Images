#include "hip/hip_runtime.h"
#include<bits/stdc++.h>
using namespace std;
#define pi (2.0*acos(0.0))
#define eps 1e-6
#define ll long long
#define inf (1<<29)
#define vi vector<int>
#define vll vector<ll>
#define sc(x) scanf("%d",&x)
#define scl(x) scanf("%lld",&x)
#define all(v) v.begin() , v.end()
#define me(a,val) memset( a , val ,sizeof(a) )
#define pb(x) push_back(x)
#define pii pair<int,int> 
#define mp(a,b) make_pair(a,b)
#define Q(x) (x) * (x)
#define L(x) ((x<<1) + 1)
#define R(x) ((x<<1) + 2)
#define M(x,y) ((x+y)>>1)
#define fi first
#define se second
#define MOD 1000000007
#define ios ios::sync_with_stdio(0)
#define TRADITIONAL 1
#define N 1024

clock_t tStart;

typedef struct MatrixStruct {
	double **data;
	int height, width;
} Matrix;

typedef struct CudaMatrixStruct {
	double *data;
	int height, width;
} CudaMatrix;


void nrerror(string s){
	printf("Numerical Recipes run-time error...\n");
	printf("%s\n",s.c_str());
	printf("...now exiting to system...\n");
	exit(1);
}

double *dvector(int nl, int nh){
	double *v;
	v = (double *) calloc((unsigned) (nh-nl+1), sizeof(double));
	if (!v) nrerror("allocation failure in dvector()");
	return v-nl;
}

float *vector(int nl, int nh){
	float *v;
	v = (float *) calloc((unsigned) (nh-nl+1), sizeof(float));
	if (!v) nrerror("allocation failure in dvector()");
	return v-nl;
}

void free_dvector(double *v, int nl, int nh){
	free((char*) (v+nl));
}

void free_vector(float *v, int nl, int nh){
	free((char*) (v+nl));
}

float **matrix(int nrl,int nrh,int ncl,int nch){
	float **m;
	m = (float **) malloc((unsigned) (nrh-nrl+1)*sizeof(float*));
	if (!m) nrerror("allocation failure 1 in matrix()");
	m -= nrl;
	for(int i = nrl ; i <= nrh ; i++){
		m[i]=(float *) malloc((unsigned) (nch-ncl+1)*sizeof(float));
		if (!m[i]) nrerror("allocation failure 2 in matrix()");
		m[i] -= ncl;
	}
	return m;
}

void free_matrix(float **m,int nrl,int nrh,int ncl,int nch){
	for(int i = nrh ; i >= nrl ; i--) free((char*) (m[i]+ncl));
	free((char*) (m+nrl));
}

double **dmatrix(int nrl, int nrh, int ncl, int nch){
	double **m;
	m = (double **) calloc((unsigned) (nrh-nrl+1), sizeof(double*));
	if (!m) nrerror("allocation failure 1 in dmatrix()");
	m -= nrl;
	for(int i = nrl ; i <= nrh ; i++){
		m[i] = (double *) calloc((unsigned) (nch-ncl+1), sizeof(double));
		if (!m[i]) nrerror("allocation failure 2 in dmatrix()");
		m[i] -= ncl;
	}
	return m;
}

void free_dmatrix(double **m, int nrl, int nrh, int ncl, int nch){
	for(int i = nrh ; i >= nrl ; i--) free((char*) (m[i]+ncl));
	free((char*) (m+nrl));
}

/*
double log2(double x){
	return log10(x) / log10( 2.0 );
}
*/

void CreateMatrix(Matrix **M, int hei, int wid){
	Matrix *tmp;

	tmp = (Matrix *) calloc(1, sizeof(Matrix)); 
	tmp->data = (double **) calloc(hei, sizeof(double *));
	
	if (!(tmp->data)) {
		nrerror("allocation failure in CreateMatrix()");
		exit(1);
	}

	for (int h = 0 ; h < hei ; h++) {
		tmp->data[h] = (double *) calloc(wid, sizeof(double));
		if (!(tmp->data[h])) {
			nrerror("allocation failure in CreateMatrix()");
			exit(1);
		}
	}

	tmp->height = hei;
	tmp->width = wid;
	*M = tmp;
}

void FreeMatrix(Matrix *M){
	int hei = M->height;
	for(int h = 0 ; h < hei ; h++){
	     free(M->data[h]);
	}
	free(M->data);
	free(M);
}


void four1(double *data, int nn, int isign){
	int n, mmax, m, j, istep, i;
	double wtemp, wr, wpr, wpi, wi, theta;
	double tempr, tempi;
	n = nn << 1;
	j = 1;
	for (i=1;i<n;i+=2) {
		if (j > i) {
			swap(data[j],data[i]);
			swap(data[j+1],data[i+1]);
		}
		m = n >> 1;
		while (m >= 2 && j > m) {
			j -= m;
			m >>= 1;
		}
		j += m;
	}
	mmax = 2;
	while (n > mmax) {
		istep = 2*mmax;
		theta = 6.28318530717959/(isign*mmax);
		wtemp = sin(0.5*theta);
		wpr = -2.0*wtemp*wtemp;
		wpi = sin(theta);
		wr = 1.0;
		wi = 0.0;
		for (m=1;m<mmax;m+=2) {
			for (i=m;i<=n;i+=istep) {
				j = i+mmax;
				tempr = wr*data[j]-wi*data[j+1];
				tempi = wr*data[j+1]+wi*data[j];
				data[j] = data[i]-tempr;
				data[j+1] = data[i+1]-tempi;
				data[i] += tempr;
				data[i+1] += tempi;
			}
			wr = (wtemp=wr)*wpr-wi*wpi+wr;
			wi = wi*wpr+wtemp*wpi+wi;
		}
		mmax = istep;
	}
}

void four2(double **fftr, double **ffti, double **rdata, double **idata, int rs, int cs, int isign){
/************************************************************ 

   2-D fourier transform of data with real part stored in
   "rdata" and imaginary part in "idata" with size "rs" x
   "cs". The result is in "fftr" and "ffti". The isign is
   "isign" =  1 forward, and "isign" = -1 inverse 

*************************************************************/
        double **T, *tmp1, *tmp2;
        int i, j;

        tmp1 = dvector(1,2*cs);
        tmp2 = dvector(1,2*rs);
        T = dmatrix(1,2*rs,1,cs);

        for (i=1;i<=rs;i++) {
            for (j=1;j<=cs;j++) {
                tmp1[j*2-1] = rdata[i][j];
                tmp1[j*2] = idata[i][j];
            }
            four1(tmp1, cs, isign);
            for (j=1;j<=cs;j++) {
                T[i*2-1][j] = tmp1[j*2-1];
                T[i*2][j] = tmp1[j*2];
            }
        }

        for (i=1;i<=cs;i++) {
            for (j=1;j<=rs;j++) {
                tmp2[j*2-1] = T[j*2-1][i];
                tmp2[j*2] = T[j*2][i];
            }
            four1(tmp2,rs,isign);
            for (j=1;j<=rs;j++) {
                fftr[j][i] = tmp2[j*2-1];
                ffti[j][i] = tmp2[j*2];
            }
        }
        free_dvector(tmp1, 1, 2*cs);
        free_dvector(tmp2, 1, 2*rs);
        free_dmatrix(T, 1, 2*rs, 1, cs); 
}

void Mat_FFT2(Matrix *Output_real, Matrix *Output_imag, Matrix *Input_real, Matrix *Input_imag){
	int xs, ys, i, j;
	double **R, **I, **Fr, **Fi;

	xs = Input_real->height;
	ys = Input_real->width;

    R  = dmatrix(1,xs,1,ys);
    I  = dmatrix(1,xs,1,ys);
    Fr = dmatrix(1,xs,1,ys);
    Fi = dmatrix(1,xs,1,ys);
		
    for (i=1;i<=Input_real->height;i++) 
        for (j=1;j<=Input_real->width;j++) {
            R[i][j] = Input_real->data[i-1][j-1];
            I[i][j] = Input_imag->data[i-1][j-1];
        }

    four2(Fr, Fi, R, I, xs, ys, 1);         /* 2-D FFT */

    for (i=1;i<=Input_real->height;i++) 
        for (j=1;j<=Input_real->width;j++) {
            Output_real->data[i-1][j-1] = Fr[i][j];
            Output_imag->data[i-1][j-1] = Fi[i][j];
        }

    free_dmatrix(R,1,xs,1,ys);
    free_dmatrix(I,1,xs,1,ys);   
    free_dmatrix(Fr,1,xs,1,ys);
    free_dmatrix(Fi,1,xs,1,ys);   
}

void Mat_IFFT2(Matrix *Output_real, Matrix *Output_imag, Matrix *Input_real, Matrix *Input_imag){
	int xs, ys, i, j;
	double **R, **I, **Fr, **Fi, NN;

	xs = Input_real->height;
	ys = Input_real->width;

    R  = dmatrix(1,xs,1,ys);
    I  = dmatrix(1,xs,1,ys);
    Fr = dmatrix(1,xs,1,ys);
    Fi = dmatrix(1,xs,1,ys);

    for (i=1;i<=Input_real->height;i++) 
        for (j=1;j<=Input_real->width;j++) {
            R[i][j] = Input_real->data[i-1][j-1];
            I[i][j] = Input_imag->data[i-1][j-1];
        }

    four2(Fr, Fi, R, I, xs, ys, -1);         /* 2-D IFFT */

	NN = (double) (xs*ys);

    for (i=1;i<=Input_real->height;i++) 
        for (j=1;j<=Input_real->width;j++) {
            Output_real->data[i-1][j-1] = Fr[i][j]/NN;
            Output_imag->data[i-1][j-1] = Fi[i][j]/NN;
        }

    free_dmatrix(R,1,xs,1,ys);
    free_dmatrix(I,1,xs,1,ys);   
    free_dmatrix(Fr,1,xs,1,ys);
    free_dmatrix(Fi,1,xs,1,ys);   
}

void Mat_Copy(Matrix *A, Matrix *B, int h_target, int w_target, int h_begin, int w_begin, int h_end, int w_end){
	int i, j, h, w, h_done, w_done;
	if ((h_target >= 0)&&(h_target < A->height)&&(w_target >= 0)&&(w_target < A->width)) {
		if ((h_begin >= 0)&&(h_begin < B->height)&&(w_begin >= 0)&&(w_begin < B->width)) {
			h = h_end-h_begin+1;
			w = w_end-w_begin+1;
			if ((h >= 1)&&(w >= 1)) {
				h_done = h_target+h-1;
				w_done = w_target+w-1;
				if ((h_done < A->height)&&(w_done < A->width)) {
					for (i=0;i<h;i++) {
						for (j=0;j<w;j++) {
							A->data[i+h_target][j+w_target] = B->data[i+h_begin][j+w_begin];
						}
					}
				}
			}
		}
	}
	else {
		printf("matrix dimension error!\n");
		exit(1);
	}
}


void Mat_Product(Matrix *A, Matrix *B, Matrix *C){
	for(int h = 0 ; h < A->height ; h++)
		for(int w = 0 ; w < A->width ; w++)
			A->data[h][w] = B->data[h][w]*C->data[h][w];
}

void Mat_Sum(Matrix *A, Matrix *B, Matrix *C){
	for(int h = 0 ; h < A-> height ; h++)
		for(int w = 0 ; w < A->width ; w++)
			A->data[h][w] = B->data[h][w]+C->data[h][w];
}

void Mat_Substract(Matrix *A, Matrix *B, Matrix *C){
	for(int h = 0 ; h < A->height ; h++)
		for(int w = 0 ; w < A->width ; w++)
			A->data[h][w] = B->data[h][w]-C->data[h][w];
}

void Gabor(Matrix *Gr, Matrix *Gi, int s, int n, double Ul, double Uh, int scale, int orientation, int flag);

__device__ void CudaPrintMatrix(CudaMatrix *A){
	printf("CudaMatrix->height: %d  CudaMatrix->width: %d\n", A->height, A->width);
	printf("Elementos de CudaMatrix:\n");
	for(int i = 0; i < A->height * A->width; i++)
		if(fabs(A->data[i]) > 1e-6) printf("ERROR: %.1lf ", A->data[i]);
	printf("\n");	
}

void PrintMatrix(Matrix *A){
	printf("Matrix->height: %d  Matrix->width: %d\n", A->height, A->width);
	printf("Elementos de Matrix:\n");
	for(int i = 0; i < A->height ; i++)
		for(int j = 0 ; j < A->width ; j++)
			printf("%.1lf ", A->data[i][j] );
	printf("\n");		
}

void CudaCreateMatrix(CudaMatrix *&AA,int hei,int wid){	
	CudaMatrix *A = new CudaMatrix();
	A->width = wid;
	A->height = hei;

	hipMalloc((void **)&AA, sizeof(CudaMatrix));
	double * data;
	hipMalloc((void **) &data, sizeof(double) * hei * wid);

	hipMemcpy(AA, A, sizeof(CudaMatrix), hipMemcpyHostToDevice);
	hipMemcpy(&(AA->data), &data, sizeof(double *), hipMemcpyHostToDevice);
}

__global__ void PrintD(double *dev, int a){
	
	/*for(int i = 0 ; i < a ; i++)
		if(fabs(dev[i]) > 1e-6) printf("tmr: %.1lf ",dev[i]);
		
	printf("\n");*/
}

__global__ void Print(CudaMatrix *dev){
	printf("%d - %d\n",dev->height,dev->width);
	for(int i = 0 ; i < dev->height * dev->width ; i++)
		if(fabs(dev->data[i]) > 1e-6) printf("HORROR2!!: %.1lf ",dev->data[i]);
		
	printf("\n");
}

void CudaCopyMatrix(CudaMatrix *&AA,Matrix *B, int flag = 0){
	CudaMatrix *A = new CudaMatrix();
	A->width = B->width;
	A->height = B->height;

	hipMalloc((void **)&AA, sizeof(CudaMatrix));
	double *data , *dataB;
	dataB = (double *)malloc( sizeof(double) * B->height * B->width );
	for(int i = 0 ; i < B->height ; i++)
		for(int j = 0 ; j < B->width ; j++)
			dataB[ i * B->width + j ] = B->data[i][j];
	//if(flag){	
		//for(int i = 1000; i <  2000; i++) printf("%.1lf ", dataB[i]);
		//printf("\n"); 
	//}
	
	hipMalloc((void **) &data, sizeof(double) * B->height * B->width );
	hipMemcpy( data , dataB , sizeof(double) * B->height * B->width , hipMemcpyHostToDevice );
	
	//if(flag)PrintD<<<1,1>>>(data, B->height * B->width);
	
	/*double * data2;
	data2 = (double *)malloc( sizeof(double) * B->height * B->width );
	hipMemcpy( data2 , data , sizeof(double) * B->height * B->width , hipMemcpyDeviceToHost );
	
	for(int i = 0; i < B->height * B->width ; i++) if(fabs(data2[i] - dataB[i]) > 1e-6) printf("HORROR!! ");
	*/
	//hipFree(data);
	
	hipMemcpy(AA, A, sizeof(CudaMatrix), hipMemcpyHostToDevice);
	hipMemcpy(&(AA->data), &data, sizeof(double *), hipMemcpyHostToDevice);
	
	//if(flag) Print<<<1,1>>>(AA);
	
	//hipFree(AA);
}

__device__ void CudaGabor(CudaMatrix *Gr, CudaMatrix *Gi, int s, int n, double Ul, double Uh, int scale, int orientation, int flag){
	double base, a, u0, z, Uvar, Vvar, Xvar, Yvar, X, Y, G, t1, t2, m;
	int x, y, side;

	base = Uh/Ul;
	a = pow(base, 1.0/(double)(scale-1));

	u0 = Uh/pow(a, (double) scale-s);

	Uvar = (a-1.0)*u0/((a+1.0)*sqrt(2.0*log(2.0)));

	z = -2.0*log(2.0)*(Uvar*Uvar)/u0;
	Vvar = tan(pi/(2*orientation))*(u0+z)/sqrt(2.0*log(2.0)-z*z/(Uvar*Uvar));

        Xvar = 1.0/(2.0*pi*Uvar);
        Yvar = 1.0/(2.0*pi*Vvar);

	t1 = cos(pi/orientation*(n-1.0));
	t2 = sin(pi/orientation*(n-1.0));

	side = (int) (Gr->height-1)/2;

	//CUDA
	for (x=0;x<2*side+1;x++) {
		for (y=0;y<2*side+1;y++) {
			X = (double) (x-side)*t1+ (double) (y-side)*t2;
			Y = (double) -(x-side)*t2+ (double) (y-side)*t1;
			G = 1.0/(2.0*pi*Xvar*Yvar)*pow(a, (double) scale-s)*exp(-0.5*((X*X)/(Xvar*Xvar)+(Y*Y)/(Yvar*Yvar)));
			Gr->data[x * (2*side+1) + y] = G*cos(2.0*pi*u0*X);
			Gi->data[x * (2*side+1) + y] = G*sin(2.0*pi*u0*X);
		}
	}

	/* if flag = 1, then remove the DC from the filter */
	

	if (flag == 1) {
	
		//CUDA - logn
		m = 0;
		for (x=0;x<2*side+1;x++)
			for (y=0;y<2*side+1;y++)
				m += Gr->data[x * (2*side+1) + y];

		m /= pow((double) 2.0*side+1, 2.0);
		
		
		//CUDA
		for (x=0;x<2*side+1;x++)
			for (y=0;y<2*side+1;y++)
				Gr->data[x * (2*side+1) + y] -= m;
	}	
}

__device__ void CudaMat_Copy(CudaMatrix *A, CudaMatrix *B, int h_target, int w_target, int h_begin, int w_begin, int h_end, int w_end){
	int i, j, h, w, h_done, w_done;
	if ((h_target >= 0)&&(h_target < A->height)&&(w_target >= 0)&&(w_target < A->width)) {
		if ((h_begin >= 0)&&(h_begin < B->height)&&(w_begin >= 0)&&(w_begin < B->width)) {
			h = h_end-h_begin+1;
			w = w_end-w_begin+1;
			if ((h >= 1)&&(w >= 1)) {
				h_done = h_target+h-1;
				w_done = w_target+w-1;
				if ((h_done < A->height)&&(w_done < A->width)) {
					for (i=0;i<h;i++) {
						for (j=0;j<w;j++) {
						
							A->data[(i+h_target) * w + (j+w_target)] = B->data[(i+h_begin) * w + (j+w_begin)];
						}
					}
				}
			}
		}
	}
	else {
		printf("matrix dimension error!\n");
		//exit(1);
	}
}

__device__  void CudaMat_Product(CudaMatrix *A, CudaMatrix *B, CudaMatrix *C){
	for(int h = 0 ; h < A->height ; h++)
		for(int w = 0 ; w < A->width ; w++)
			A->data[h * A->width + w] = B->data[h * A->width + w] * C->data[h * A->width + w];
}

__device__  void CudaMat_Sum(CudaMatrix *A, CudaMatrix *B, CudaMatrix *C){
	for(int h = 0 ; h < A-> height ; h++)
		for(int w = 0 ; w < A->width ; w++)
			A->data[h * A->width + w] = B->data[h * A->width + w] + C->data[h * A->width + w];
}

__device__  void CudaMat_Substract(CudaMatrix *A, CudaMatrix *B, CudaMatrix *C){
	for(int h = 0 ; h < A->height ; h++)
		for(int w = 0 ; w < A->width ; w++)
			A->data[h * A->width + w] = B->data[h * A->width + w] - C->data[h * A->width + w];
}

__device__ void Cudadmatrix(double *&A, int nrl, int nrh, int ncl, int nch){
	A = (double *)malloc( sizeof(double) * (nrh + 1) * (nch + 1) );
	
	//A = new double[(nrh + 1) * (nch + 1)];
}

__device__ void Cudafree_dmatrix(double *&m, int nrl, int nrh, int ncl, int nch){
	//hipFree(m);
	
	free(m);
}

__device__ void Cudadvector(double * &A, int nl, int nh){
	A = (double *)malloc( sizeof(double) * (nh  + 1) );
	//A = new double[nh + 1];

}

__device__ void Cudafree_dvector(double *&v, int nl, int nh){
	free(v);
}

__device__ inline void Cudaswap(double &x, double &y){
	double tmp;
	tmp = y; y = x; x = tmp;
}
__device__ void Cudafour1(double *data, int nn, int isign){
	int n, mmax, m, j, istep, i;
	double wtemp, wr, wpr, wpi, wi, theta;
	double tempr, tempi;
	
	printf("dunk 1\n");
	n = nn << 1;
	j = 1;
	for (i=1;i<n;i+=2) {
		if (j > i) {
			Cudaswap(data[j],data[i]);
			Cudaswap(data[j+1],data[i+1]);
		}
		m = n >> 1;
		while (m >= 2 && j > m) {
			j -= m;
			m >>= 1;
		}
		j += m;
	}
	mmax = 2;
	while (n > mmax) {
		istep = 2*mmax;
		theta = 6.28318530717959/(isign*mmax);
		wtemp = sin(0.5*theta);
		wpr = -2.0*wtemp*wtemp;
		wpi = sin(theta);
		wr = 1.0;
		wi = 0.0;
		for (m=1;m<mmax;m+=2) {
			for (i=m;i<=n;i+=istep) {
				j = i+mmax;
				tempr = wr*data[j]-wi*data[j+1];
				tempi = wr*data[j+1]+wi*data[j];
				data[j] = data[i]-tempr;
				data[j+1] = data[i+1]-tempi;
				data[i] += tempr;
				data[i+1] += tempi;
			}
			wr = (wtemp=wr)*wpr-wi*wpi+wr;
			wi = wi*wpr+wtemp*wpi+wi;
		}
		mmax = istep;
	}
	printf("dunk 2\n");
}

__device__ void Cudafour2(double *fftr, double *ffti, double *rdata, double *idata, int rs, int cs, int isign){
/************************************************************ 

   2-D fourier transform of data with real part stored in
   "rdata" and imaginary part in "idata" with size "rs" x
   "cs". The result is in "fftr" and "ffti". The isign is
   "isign" =  1 forward, and "isign" = -1 inverse 

*************************************************************/
        double *T, *tmp1, *tmp2;
        int i, j;
		
		printf("this 1\n");
        Cudadvector(tmp1, 1,2*cs);
        Cudadvector(tmp2, 1,2*rs);

        Cudadmatrix(T, 1,2*rs,1,cs);
		
		for(int i = 0 ; i < 10 ; i++)
			T[i] = 
		
		printf("\n");
		
        for (i=1;i<=rs;i++) {
            for (j=1;j<=cs;j++) {
                tmp1[j*2-1] = rdata[i * (cs + 1) + j];
                tmp1[j*2] = idata[i * (cs + 1) + j];
            }
            printf("this 2\n");
            Cudafour1(tmp1, cs, isign);
            printf("this 3\n");
            for (j=1;j<=cs;j++){
            	printf("-> %d\n",j);
            	printf("mul: %d %.5lf\n", (i*2) * (cs + 1) + j , T[0] );
                T[(i*2-1) * (cs + 1) + j] = tmp1[j*2-1];
                T[(i*2) * (cs + 1) + j] = tmp1[j*2];
            }
        }
        printf("this 4\n");

        for (i=1;i<=cs;i++) {
            for (j=1;j<=rs;j++) {
                tmp2[j*2-1] = T[(j*2-1) * (cs + 1) + i];
                tmp2[j*2] = T[(j*2) * (cs + 1) + i];
            }
            Cudafour1(tmp2,rs,isign);
            for (j=1;j<=rs;j++) {
                fftr[j * (cs + 1) + i] = tmp2[j*2-1];
                ffti[j * (cs + 1) + i] = tmp2[j*2];
            }
        }
        
        Cudafree_dvector(tmp1, 1, 2*cs);
        Cudafree_dvector(tmp2, 1, 2*rs);
        Cudafree_dmatrix(T, 1, 2*rs, 1, cs); 
}

__device__ void CudaMat_FFT2(CudaMatrix *Output_real, CudaMatrix *Output_imag, CudaMatrix *Input_real, CudaMatrix *Input_imag){
	int xs, ys, i, j;
	//double **R, **I, **Fr, **Fi;
	double *R, *I, *Fr, *Fi;

	xs = Input_real->height;
	ys = Input_real->width;
	
	printf("here 1\n");
    Cudadmatrix(R, 1,xs,1,ys);
    Cudadmatrix(I, 1,xs,1,ys);
    Cudadmatrix(Fr, 1,xs,1,ys);
    Cudadmatrix(Fi, 1,xs,1,ys);
	
	
    for (i=1;i<=Input_real->height;i++) 
        for (j=1;j<=Input_real->width;j++) {
            R[i * (Input_real->width + 1) + j] = Input_real->data[(i-1) * Input_real->width + (j-1)];
            I[i * (Input_real->width + 1) + j] = Input_imag->data[(i-1) * Input_real->width + (j-1)];
        }
	
	
    Cudafour2(Fr, Fi, R, I, xs, ys, 1);         /* 2-D FFT */
	
	printf("here 2\n");
	
    for (i=1;i<=Input_real->height;i++) 
        for (j=1;j<=Input_real->width;j++) {
            Output_real->data[(i-1) * Input_real->width + (j-1)] = Fr[i * (Input_real->width + 1) + j];
            Output_imag->data[(i-1) * Input_real->width + (j-1)] = Fi[i * (Input_real->width + 1) + j];
        }

    Cudafree_dmatrix(R,1,xs,1,ys);
    Cudafree_dmatrix(I,1,xs,1,ys);   
    Cudafree_dmatrix(Fr,1,xs,1,ys);
    Cudafree_dmatrix(Fi,1,xs,1,ys);   
}

__device__ void CudaMat_IFFT2(CudaMatrix *Output_real, CudaMatrix *Output_imag, CudaMatrix *Input_real, CudaMatrix *Input_imag){
	int xs, ys, i, j;
	double *R, *I, *Fr, *Fi, NN;

	xs = Input_real->height;
	ys = Input_real->width;

    Cudadmatrix(R, 1,xs,1,ys);
    Cudadmatrix(I, 1,xs,1,ys);
    Cudadmatrix(Fr, 1,xs,1,ys);
    Cudadmatrix(Fi, 1,xs,1,ys);

    for (i=1;i<=Input_real->height;i++) 
        for (j=1;j<=Input_real->width;j++) {
            R[i * (Input_real->width + 1) + j] = Input_real->data[(i-1) * Input_real->width + (j-1)];
            I[i * (Input_real->width + 1) + j] = Input_imag->data[(i-1) * Input_real->width + (j-1)];
        }

    Cudafour2(Fr, Fi, R, I, xs, ys, -1);         /* 2-D IFFT */

	NN = (double) (xs*ys);

    for (i=1;i<=Input_real->height;i++) 
        for (j=1;j<=Input_real->width;j++) {
            Output_real->data[(i-1) * Input_real->width + (j-1)] = Fr[i * (Input_real->width + 1) + j]/NN;
            Output_imag->data[(i-1) * Input_real->width + (j-1)] = Fi[i * (Input_real->width + 1) + j]/NN;
        }

    Cudafree_dmatrix(R,1,xs,1,ys);
    Cudafree_dmatrix(I,1,xs,1,ys);   
    Cudafree_dmatrix(Fr,1,xs,1,ys);
    Cudafree_dmatrix(Fi,1,xs,1,ys);   
}

__device__ void CudaCopyMatrixValues(CudaMatrix *&A, CudaMatrix *B){
	
	A = new CudaMatrix();
	A->width = 3;//B->width;
	A->height = 3;//B->height;
	
	//A->data = new double[B->width * B->height];
	
	A->data = (double *)malloc(A->width * A->height * sizeof(double));
	
	printf("aw: %d ah: %d\n", A->width, A->height);
	
	for(int i = 0; i < A->width * A->height; i++){
		A->data[i] = B->data[i];
		if(i < 12) printf("%d: %.1lf %.1lf\n", i, A->data[i], B->data[i]);
	}
	printf("\n");
}

__global__ void f(CudaMatrix *Gr ,CudaMatrix *Gi ,double Ul ,double Uh ,int scale ,int orientation ,int flag ,CudaMatrix *F_1 ,CudaMatrix *F_2 ,int side ,
				  CudaMatrix *G_real ,CudaMatrix *G_imag ,CudaMatrix *Tmp_1 ,CudaMatrix *Tmp_2, CudaMatrix *F ,double *features , CudaMatrix *F_real,
				  CudaMatrix *F_imag , CudaMatrix *IMG , CudaMatrix *IMG_imag , int hei, int wid, CudaMatrix *FilteredImg_real, CudaMatrix *FilteredImg_imag){
				  
	int s = blockIdx.x;
	int n = threadIdx.x;
	printf("f: %d %d\n", s, n);
	
	//if( s == 1 && n == 0 ){
	/*CudaMatrix * dev_IMG;
	
	CudaCopyMatrixValues(dev_IMG, IMG);

	printf("Matrix_dev_IMG\n");
	//CudaPrintMatrix(dev_IMG);
	
	free(dev_IMG);
	//}
	*/
	
	CudaGabor(Gr, Gi, s, n, Ul, Uh, scale, orientation, flag);
	
	CudaMat_Copy(F_1, Gr, 0, 0, 0, 0, 2*side, 2*side);//CUDA
	CudaMat_Copy(F_2, Gi, 0, 0, 0, 0, 2*side, 2*side);//CUDA
	
	printf("por cua1?\n");
				
	CudaMat_FFT2(G_real, G_imag, F_1, F_2);//CUDA-no definido
	
	printf("por cua2?\n");
	
	if( s == 0 && n == 0 ){
		double ans = 0;
		for(int i = 0 ; i < F_1->height ; i++)
			for(int j = 0 ; j < F_1->width ; j++)
				ans += F_1->data[ i * F_1->width + j];
		printf("GPU ---> %.5lf\n",ans);
	}
	
	CudaMat_Product(Tmp_1, G_real, F_real);//CUDA
	CudaMat_Product(Tmp_2, G_imag, F_imag);//CUDA
	CudaMat_Substract(IMG, Tmp_1, Tmp_2);//CUDA
	
	
	CudaMat_Product(Tmp_1, G_real, F_imag);//CUDA
	CudaMat_Product(Tmp_2, G_imag, F_real);//CUDA
	CudaMat_Sum(IMG_imag, Tmp_1, Tmp_2);//CUDA
	
	CudaMat_IFFT2(Tmp_1, Tmp_2, IMG, IMG_imag);//CUDA-no definido
	
	//CUDA - logn
	int base = scale * orientation;
	
	double m = 0;
	for (int h=0;h<hei;h++)
		for (int w=0;w<wid;w++) {
			F->data[h * wid + w] = sqrt(pow(IMG->data[h * wid + w], 2.0)+pow(IMG_imag->data[h * wid + w], 2.0));
			m += F->data[h * wid + w];
		}

	m /= (double) (hei*wid);
	features[s*orientation+n] = (float) m;
	
	//CUDA - logn
	double v = 0;
	for (int h=0;h<hei;h++)
		for (int w=0;w<wid;w++)
			v += (F->data[h * wid + w]-m)*(F->data[h * wid + w]-m);
	
	v /= (double) (hei*wid);
	features[base+s*orientation+n] = (float) sqrt(v);
    
	CudaMat_Copy(FilteredImg_real, Tmp_1, s*hei, n*wid, 2*side, 2*side, hei+2*side-1, wid+2*side-1);//CUDA
	CudaMat_Copy(FilteredImg_imag, Tmp_2, s*hei, n*wid, 2*side, 2*side, hei+2*side-1, wid+2*side-1);//CUDA
	
}

void GaborFilteredImg(Matrix *FilteredImg_real, Matrix *FilteredImg_imag, Matrix *img, int side, double Ul, double Uh, int scale, int orientation, int flag){
	int h, w, xs, ys, border, r1, r2, r3, r4, hei, wid, s, n, base;
	Matrix *IMG, *IMG_imag, *Gr, *Gi, *Tmp_1, *Tmp_2, *F_1, *F_2, *G_real, *G_imag, *F_real, *F_imag,*F;
	double m, v;
	
	base = scale*orientation;
	double *features; //exact memory
	features = (double *)malloc( 2 * scale * orientation * sizeof(double) );

	border = side;
	hei = img->height;
	wid = img->width;

	/* FFT2 */
	xs = (int) pow(2.0, ceil(log2((double)(img->height+2.0*border))));
	ys = (int) pow(2.0, ceil(log2((double)(img->width+2.0*border))));

	CreateMatrix(&IMG, xs, ys);

	r1 = img->width+border;
	r2 = img->width+2*border;
	for (h=0;h<border;h++) {
		for (w=0;w<border;w++)
			IMG->data[h][w] = img->data[border-1-h][border-1-w];
		for (w=border;w<r1;w++)
			IMG->data[h][w] = img->data[border-1-h][w-border];
		for (w=r1;w<r2;w++)
			IMG->data[h][w] = img->data[border-1-h][2*img->width-w+border-1];
	}

	r1 = img->height+border;
	r2 = img->width+border;
	r3 = img->width+2*border;
	for (h=border;h<r1;h++) {
		for (w=0;w<border;w++)
			IMG->data[h][w] = img->data[h-border][border-1-w];
		for (w=border;w<r2;w++)
			IMG->data[h][w] = img->data[h-border][w-border];
		for (w=r2;w<r3;w++)
			IMG->data[h][w] = img->data[h-border][2*img->width-w+border-1];
	}

	r1 = img->height+border;
	r2 = img->height+2*border;
	r3 = img->width+border;
	r4 = img->width+2*border;
	for (h=r1;h<r2;h++) {
		for (w=0;w<border;w++)
			IMG->data[h][w] = img->data[2*img->height-h+border-1][border-1-w];
		for (w=border;w<r3;w++)
			IMG->data[h][w] = img->data[2*img->height-h+border-1][w-border];
		for (w=r3;w<r4;w++)
			IMG->data[h][w] = img->data[2*img->height-h+border-1][2*img->width-w+border-1];
	}
	
	CreateMatrix(&F_real, xs, ys);
	CreateMatrix(&F_imag, xs, ys);
	CreateMatrix(&IMG_imag, xs, ys);

	
	Mat_FFT2(F_real, F_imag, IMG, IMG_imag);
	
	//Declaring variables for CUDA process
	CudaMatrix *dev_Gr, *dev_Gi, *dev_Tmp1, *dev_Tmp2, *dev_F_1, *dev_F_2, *dev_G_real, *dev_G_imag, *dev_F, *dev_F_real , *dev_F_imag, *dev_IMG, *dev_IMG_imag, *dev_FilteredImg_real, *dev_FilteredImg_imag;
	
	//Coping all the matrix
	CudaCopyMatrix( dev_F_real , F_real );
	CudaCopyMatrix( dev_F_imag , F_imag );
	CudaCopyMatrix( dev_IMG , IMG);
	CudaCopyMatrix( dev_IMG_imag , IMG_imag );
	CudaCopyMatrix( dev_FilteredImg_real, FilteredImg_real);
	CudaCopyMatrix( dev_FilteredImg_imag, FilteredImg_imag);
	
	//Print<<<1,1>>>( dev_IMG );
	
	//Debug
	
	//PrintMatrix(IMG);
	//printf("\n\n");
	//Print<<<1,1>>>( dev_IMG );
	
	/////
	
	//Creating Matrix for CUDA and free
	CudaCreateMatrix( dev_Gr , 2 * side + 1 , 2 * side + 1 );
	CudaCreateMatrix( dev_Gi , 2 * side + 1 , 2 * side + 1 );
	CudaCreateMatrix( dev_Tmp1 , xs , ys );
	CudaCreateMatrix( dev_Tmp2 , xs , ys );
	CudaCreateMatrix( dev_F_1 , xs , ys );
	CudaCreateMatrix( dev_F_2 , xs , ys );
	CudaCreateMatrix( dev_G_real , xs , ys );
	CudaCreateMatrix( dev_G_imag , xs , ys );
	CudaCreateMatrix( dev_F , hei , wid );
	
	double *dev_features;
	hipMalloc( &dev_features , 2 * scale * orientation * sizeof(double) );
	hipMemset( (void *)dev_features , 0.0 , 2 * scale * orientation * sizeof(double) );

	f<<< 1 , 1 >>>( dev_Gr , dev_Gi , Ul , Uh , scale , orientation, flag, dev_F_1 , dev_F_2 , side , dev_G_real , dev_G_imag , dev_Tmp1 , dev_Tmp2 , dev_F ,
					dev_features , dev_F_real, dev_F_imag , dev_IMG , dev_IMG_imag , hei , wid, dev_FilteredImg_real, dev_FilteredImg_imag);
	
	double *features2 = new double [2 * scale * orientation];
	
	hipMemcpy(features2, dev_features, 2 * scale * orientation * sizeof(double), hipMemcpyDeviceToHost);
	
	for(int i = 0 ; i < 2 * scale * orientation ; i++)
		printf("%.2lf ",features2[i]);
	printf("\n");
	
	//LIBERA LA MATRIZ CREADA (de paso tambien libera "data" de la funcion creo)
	hipFree(dev_F_real);
	hipFree(dev_F_imag);
	hipFree(dev_Gr);
	hipFree(dev_Gi);
	hipFree(dev_Tmp1);
	hipFree(dev_Tmp2);
	hipFree(dev_F_1);
	hipFree(dev_F_2);
	hipFree(dev_G_real);
	hipFree(dev_G_imag);
	hipFree(dev_F);	
	hipFree(dev_features);
	hipFree(dev_FilteredImg_real);
	hipFree(dev_FilteredImg_imag);
	hipFree(dev_IMG);
	hipFree(dev_IMG_imag);
	
	/////////////////////////
	
	/* ----------- compute the Gabor filtered output ------------- */
	CreateMatrix(&Gr, 2*side+1, 2*side+1);   
	CreateMatrix(&Gi, 2*side+1, 2*side+1);
	CreateMatrix(&Tmp_1, xs, ys);
	CreateMatrix(&Tmp_2, xs, ys);
	CreateMatrix(&F_1, xs, ys);
	CreateMatrix(&F_2, xs, ys);
	CreateMatrix(&G_real, xs, ys);
	CreateMatrix(&G_imag, xs, ys);
    CreateMatrix(&F, hei, wid);
    
	for (s=0;s<scale;s++){
		for (n=0;n<orientation;n++){
			if( s != 0 or n != 0 ) continue;
			Gabor(Gr, Gi, s+1, n+1, Ul, Uh, scale, orientation, flag);//CUDA- 2 normales y logn
			
			double x = 0 , y = 0;
			for(int i = 0 ; i < Gr->height ; i++)
					for(int j = 0 ; j < Gr->width ; j++)
						x += Gr->data[i][j] , y += Gi->data[i][j];
			
			printf("CPU x and y: %.25lf %.25lf\n", x , y );
			
			Mat_Copy(F_1, Gr, 0, 0, 0, 0, 2*side, 2*side);//CUDA
			Mat_Copy(F_2, Gi, 0, 0, 0, 0, 2*side, 2*side);//CUDA
				
			double ac_F_1 = 0;
			for(int i = 0; i <= 2 * side; i++)
				for(int j = 0; j <= 2 * side; j++)
					ac_F_1 += F_1->data[i][j];
	
			printf("CPU ac_F_1: %.40lf\n", ac_F_1);	
			
			Mat_FFT2(G_real, G_imag, F_1, F_2);//CUDA-no definido
			
			double ans = 0;
			for(int i = 0 ; i < F_1->height ; i++)
				for(int j = 0 ; j < F_1->width ; j++)
					ans += F_1->data[i][j];
			printf("CPU ---> %.5lf\n",ans);
			
			
			Mat_Product(Tmp_1, G_real, F_real);//CUDA
			Mat_Product(Tmp_2, G_imag, F_imag);//CUDA
			Mat_Substract(IMG, Tmp_1, Tmp_2);//CUDA

			Mat_Product(Tmp_1, G_real, F_imag);//CUDA
			Mat_Product(Tmp_2, G_imag, F_real);//CUDA
			Mat_Sum(IMG_imag, Tmp_1, Tmp_2);//CUDA

			Mat_IFFT2(Tmp_1, Tmp_2, IMG, IMG_imag);//CUDA-no definido
			
			if( s == 0 && n == 0 ){
				double im = 0;
				for(int i = 0 ; i < IMG->height ; i++)
				for(int j = 0 ; j < IMG->width ; j++)
					im += fabs(IMG->data[i][j]);

				printf("CPU im: %.50lf\n", im );
			
				double ac_Tmp_2 = 0;
				for(int i = 0 ; i < xs ; i++)
					for(int j = 0 ; j < ys ; j++)
						ac_Tmp_2 += Tmp_2->data[i][j];
			
				printf("CPU ac_Tmp_2: %.50lf\n", ac_Tmp_2 );
			}
			//CUDA - logn
			m = 0;
			for (h=0;h<hei;h++)
				for (w=0;w<wid;w++) {
					F->data[h][w] = sqrt(pow(IMG->data[h][w], 2.0)+pow(IMG_imag->data[h][w], 2.0));
					m += F->data[h][w];
				}

			m /= (double) (hei*wid);
			features[s*orientation+n] = (float) m;
			
			//CUDA - logn
			v = 0;
			for (h=0;h<hei;h++)
				for (w=0;w<wid;w++)
					v += (F->data[h][w]-m)*(F->data[h][w]-m);

			v /= (double) (hei*wid);
			features[base+s*orientation+n] = (float) sqrt(v);
            
            if( s == 0 && n == 0 ) printf("CPU m: %.20lf  v: %.20lf\n",m ,v); 
            
			Mat_Copy(FilteredImg_real, Tmp_1, s*hei, n*wid, 2*side, 2*side, hei+2*side-1, wid+2*side-1);//CUDA
			Mat_Copy(FilteredImg_imag, Tmp_2, s*hei, n*wid, 2*side, 2*side, hei+2*side-1, wid+2*side-1);//CUDA
			
		}
	}
	
	for(int i = 0 ; i < 2 * scale * orientation ; i++)
		printf("%.2lf ",features[i]);
	printf("\n");

	FreeMatrix(Gr);
	FreeMatrix(Gi);
	FreeMatrix(Tmp_1);
	FreeMatrix(Tmp_2);
	FreeMatrix(F_1);
	FreeMatrix(F_2);
	FreeMatrix(G_real);
	FreeMatrix(G_imag);
	FreeMatrix(F_real);
	FreeMatrix(F_imag);
	FreeMatrix(IMG);
	FreeMatrix(IMG_imag);
}

/* ------------------------------------------------------------------------------------------------------
The Gabor function generates a Gabor filter with the selected index 's' and 'n' (scale and orientation, 
respectively) from a Gabor filter bank. This filter bank is designed by giving the range of spatial 
frequency (Uh and Ul) and the total number of scales and orientations used to partition the spectrum. 

The returned filter is stored in 'Gr' (real part) and 'Gi' (imaginary part).
--------------------------------------------------------------------------------------------------------*/
void Gabor(Matrix *Gr, Matrix *Gi, int s, int n, double Ul, double Uh, int scale, int orientation, int flag){
	double base, a, u0, z, Uvar, Vvar, Xvar, Yvar, X, Y, G, t1, t2, m;
	int x, y, side;

	base = Uh/Ul;
	a = pow(base, 1.0/(double)(scale-1));

	u0 = Uh/pow(a, (double) scale-s);

	Uvar = (a-1.0)*u0/((a+1.0)*sqrt(2.0*log(2.0)));

	z = -2.0*log(2.0)*(Uvar*Uvar)/u0;
	Vvar = tan(pi/(2*orientation))*(u0+z)/sqrt(2.0*log(2.0)-z*z/(Uvar*Uvar));

        Xvar = 1.0/(2.0*pi*Uvar);
        Yvar = 1.0/(2.0*pi*Vvar);

	t1 = cos(pi/orientation*(n-1.0));
	t2 = sin(pi/orientation*(n-1.0));

	side = (int) (Gr->height-1)/2;

	//CUDA
	for (x=0;x<2*side+1;x++) {
		for (y=0;y<2*side+1;y++) {
			X = (double) (x-side)*t1+ (double) (y-side)*t2;
			Y = (double) -(x-side)*t2+ (double) (y-side)*t1;
			G = 1.0/(2.0*pi*Xvar*Yvar)*pow(a, (double) scale-s)*exp(-0.5*((X*X)/(Xvar*Xvar)+(Y*Y)/(Yvar*Yvar)));
			Gr->data[x][y] = G*cos(2.0*pi*u0*X);
			Gi->data[x][y] = G*sin(2.0*pi*u0*X);
		}
	}

	/* if flag = 1, then remove the DC from the filter */
	

	if (flag == 1) {
	
		//CUDA - logn
		m = 0;
		for (x=0;x<2*side+1;x++)
			for (y=0;y<2*side+1;y++)
				m += Gr->data[x][y];

		m /= pow((double) 2.0*side+1, 2.0);
		
		
		//CUDA
		for (x=0;x<2*side+1;x++)
			for (y=0;y<2*side+1;y++)
				Gr->data[x][y] -= m;
	}	
}

int main(int argc, char **argv){
	
	tStart = clock();

	int hei, wid, side, scale, orientation, flag;//, s, n;
	//Matrix *Gabor_r, *Gabor_i, *Gr, *Gi, *img, *F_r, *F_i;
	Matrix *img , *F_r , *F_i;
	FILE *fp;
	unsigned char *tmp; 
	//float *output;
	double Ul, Uh;
	/* --------------------------- Example --------------------------------
		scale = 3, 
		orientation = 4, 
		Uh (highest spatial frequency) = 0.4, 
		Ul (lowest spatial frequency) = 0.1,
		flag (removing the DC term) = 0 (False),
		side (filter dimension = (2*side+1)*(2*side+1)) = 60
	----------------------------------------------------------------------- */
	scale = 3;
	orientation = 4;
	Ul = 0.1;
	Uh = 0.4;
	flag = 0;
	side = 60;

	if (argc != 4) {
		printf("usage: %s <image_name> <height> <width>\n",argv[0]);
		exit(0);
	}
	
	hei = atoi(argv[2]);
	wid = atoi(argv[3]);

	tmp = (unsigned char *) calloc(hei*wid, sizeof(unsigned char));

	if ((fp = fopen(argv[1],"r")) == NULL) {
		printf("%s can not be open!\n", argv[1]);
		exit(0);
	}
	fread(tmp, sizeof(unsigned char), hei*wid, fp);
	fclose(fp);

	
	CreateMatrix(&img, hei, wid);
	for(int i = 0 ; i < hei ; i++)
		for(int j = 0 ; j < wid ; j++)
			img->data[i][j] = (double) (tmp[i*wid+j]);
			
	free(tmp);
		
	CreateMatrix(&F_r, hei*scale, wid*orientation);
	CreateMatrix(&F_i, hei*scale, wid*orientation);

	printf("Time taken_before_Gabor: %.2fms\n", 1000.0 * (double)(clock() - tStart)/CLOCKS_PER_SEC);

	GaborFilteredImg(F_r, F_i, img, side, Ul, Uh, scale, orientation, flag);

	printf("Time taken_after_Gabor: %.2fms\n", 1000.0 * (double)(clock() - tStart)/CLOCKS_PER_SEC);	

	FreeMatrix(F_r);
	FreeMatrix(F_i);
	
	printf("Total time taken: %.2fms\n", 1000.0 * (double)(clock() - tStart)/CLOCKS_PER_SEC);
	
	return 0;
}
