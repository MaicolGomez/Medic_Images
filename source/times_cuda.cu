
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;
#define pi (2.0*acos(0.0))
#define eps 1e-6
#define ll long long
#define inf (1<<29)
#define vi vector<int>
#define vll vector<ll>
#define sc(x) scanf("%d",&x)
#define scl(x) scanf("%lld",&x)
#define all(v) v.begin() , v.end()
#define me(a,val) memset( a , val ,sizeof(a) )
#define pb(x) push_back(x)
#define pii pair<int,int> 
#define mp(a,b) make_pair(a,b)
#define Q(x) (x) * (x)
#define L(x) ((x<<1) + 1)
#define R(x) ((x<<1) + 2)
#define M(x,y) ((x+y)>>1)
#define fi first
#define se second
#define MOD 1000000007
#define ios ios::sync_with_stdio(0)
#define N 1024
#define TB 32

/*__global__ void suma(int *A, int *S){
	
	S[0] = S[0] + A[threadIdx.x];
	printf("A[t]: %d S[0]: %d\n",A[threadIdx.x], S[0]);
	
	__syncthreads();
}*/

__global__ void MatrixMultiplication(int *A,int *B,int *C){
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	
	C[ row * N + col ] = A[ row * N + col ] * B[ row * N + col ];
}

int main(){

	clock_t tStart = clock();

	int *a , *b , *c;
	size_t size = N * N * sizeof(int) ;
	a = (int *)malloc( size );
	b = (int *)malloc( size );
	c = (int *)malloc( size );
	
	for(int i = 0; i < N; i++)
		for(int j = 0; j < N; j++)
			a[i * N + j] = i + j;
			

	for(int i = 0; i < N; i++)
		for(int j = 0; j < N; j++)
			b[i * N + j] = 1;
	
	int *A , *B , *C;
	
	hipMalloc( &A , size );
	hipMalloc( &B , size );
	hipMalloc( &C , size );
	
	dim3 threadsxblock( TB , TB );
	dim3 blocksxgrid( N / threadsxblock.x , N / threadsxblock.x );
	
	hipMemcpy( A , a , size , hipMemcpyHostToDevice );
	hipMemcpy( B , b , size , hipMemcpyHostToDevice );
	
	MatrixMultiplication<<< blocksxgrid , threadsxblock >>>( A , B , C );
	
	hipMemcpy( c , C , size , hipMemcpyDeviceToHost );
	
	hipFree(A);
	hipFree(B);
	hipFree(C);
	
	printf("Time taken: %.2fms\n", 1000.0 * (double)(clock() - tStart)/CLOCKS_PER_SEC);
	
	return 0;
	
}
