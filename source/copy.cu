
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;
#define pi (2.0*acos(0.0))
#define eps 1e-6
#define ll long long
#define inf (1<<29)
#define vi vector<int>
#define vll vector<ll>
#define sc(x) scanf("%d",&x)
#define scl(x) scanf("%lld",&x)
#define all(v) v.begin() , v.end()
#define me(a,val) memset( a , val ,sizeof(a) )
#define pb(x) push_back(x)
#define pii pair<int,int> 
#define mp(a,b) make_pair(a,b)
#define Q(x) (x) * (x)
#define L(x) ((x<<1) + 1)
#define R(x) ((x<<1) + 2)
#define M(x,y) ((x+y)>>1)
#define fi first
#define se second
#define MOD 1000000007
#define ios ios::sync_with_stdio(0)


typedef struct StructA {
    int *a;
} CudaMatrix;

#define N 10

__global__ void kernel(CudaMatrix *A){
	int x = threadIdx.x;
	printf("--> %d\n",x);
	for(int i = 0 ; i < 10 ; i++)
		printf("%d - ",A->a[i]);
	printf("\n");
}

int main() {
	
    CudaMatrix *A;
    int *a;
    a = (int *)malloc( N * sizeof(int) );
    for(int i = 0 ; i < N ; i++)
    	a[i] = i;
    
    int *a2;
    hipMalloc( &a2 , N * sizeof(int) );
    hipMemcpy( a2 , a , N * sizeof(int) , hipMemcpyHostToDevice );
    
    int sz = sizeof(CudaMatrix);
	hipMalloc( &A , sz );
	
	hipMemcpy( &(A->a) , &a2 , sizeof(int *) , hipMemcpyHostToDevice );
	kernel<<<1,1>>>( A );
	
	
	hipFree( A );
}
