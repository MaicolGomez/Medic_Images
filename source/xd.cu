
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

using namespace std;

#define BLOCK_SIZE 32

__global__ void gpuMM(float *A, float *B, float *C, int N)
{
	// Matrix multiplication for NxN matrices C=A*B
	// Each thread computes a single element of C
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;

	printf("row: %d col: %d\n", col, row);

	float sum = 0.f;
	for (int n = 0; n < N; ++n)
	    sum += A[row*N+n]*B[n*N+col];

	C[row*N+col] = sum;
}

int main(int argc, char *argv[])
{
	// Perform matrix multiplication C = A*B
	// where A, B and C are NxN matrices
	// Restricted to matrices where N = K*BLOCK_SIZE;
	int N,K;
	K = 4;
	N = K*BLOCK_SIZE;
	
	cout << "Executing Matrix Multiplcation" << endl;
	cout << "Matrix size: " << N << "x" << N << endl;

	// Allocate memory on the host
	float *hA,*hB,*hC;
	hA = new float[N*N];
	hB = new float[N*N];
	hC = new float[N*N];

	// Initialize matrices on the host
	for (int j=0; j<N; j++){
	    for (int i=0; i<N; i++){
	    	hA[j*N+i] = 2.f*(j+i);
			hB[j*N+i] = 1.f*(j-i);
	    }
	}

	// Allocate memory on the device
	int size = N*N*sizeof(float);	// Size of the memory in bytes
	float *dA,*dB,*dC;
	hipMalloc(&dA,size);
	hipMalloc(&dB,size);
	hipMalloc(&dC,size);

	dim3 threadBlock(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid(K,K);
	
	// Copy matrices from the host to device
	hipMemcpy(dA,hA,size,hipMemcpyHostToDevice);
	hipMemcpy(dB,hB,size,hipMemcpyHostToDevice);
	
	//Execute the matrix multiplication kernel
	
	gpuMM<<<grid,threadBlock>>>(dA,dB,dC,N);
		
	// Now do the matrix multiplication on the CPU
	float sum;
	for (int row=0; row<N; row++){
		for (int col=0; col<N; col++){
			sum = 0.f;
			for (int n=0; n<N; n++){
				sum += hA[row*N+n]*hB[n*N+col];
			}
			hC[row*N+col] = sum;
		}
	}
	
	// Allocate memory to store the GPU answer on the host
	float *C;
	C = new float[N*N];
	
	// Now copy the GPU result back to CPU
	hipMemcpy(C,dC,size,hipMemcpyDeviceToHost);
	
	// Check the result and make sure it is correct
	for (int row=0; row<N; row++){
		for (int col=0; col<N; col++){
			if ( C[row*N+col] != hC[row*N+col] ){
				cout << "Wrong answer!" << endl;
				row = col = N;
			}
		}
	}
		
	cout << "Finished." << endl;
	
}
