/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;
#define pi (2.0*acos(0.0))
#define eps 1e-6
#define ll long long
#define inf (1<<29)
#define vi vector<int>
#define vll vector<ll>
#define sc(x) scanf("%d",&x)
#define scl(x) scanf("%lld",&x)
#define all(v) v.begin() , v.end()
#define me(a,val) memset( a , val ,sizeof(a) )
#define pb(x) push_back(x)
#define pii pair<int,int> 
#define mp(a,b) make_pair(a,b)
#define Q(x) (x) * (x)
#define L(x) ((x<<1) + 1)
#define R(x) ((x<<1) + 2)
#define M(x,y) ((x+y)>>1)
#define fi first
#define se second
#define MOD 1000000007
#define ios ios::sync_with_stdio(0)


typedef struct {
	int width;
	int height;
	float* elements;
} Matrix;

// Thread block size
#define BLOCK_SIZE 128
#define N 2048
// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);
// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE

void print(Matrix A){
	for(int i = 0 ; i < 10 ; i++){
		for(int j = 0 ; j < 10 ; j++)
			printf("%.0lf ",A.elements[ i* N + j ]);
		printf("\n");
	}
}

void MatMul(const Matrix A, const Matrix B, Matrix C){
	
	// Load A and B to device memory
	Matrix d_A;
	d_A.width = A.width; d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipMalloc(&d_A.elements, size);
	hipMemcpy(d_A.elements, A.elements, size,
	hipMemcpyHostToDevice);
	Matrix d_B;
	d_B.width = B.width; d_B.height = B.height;
	size = B.width * B.height * sizeof(float);
	hipMalloc(&d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size,
	hipMemcpyHostToDevice);
	// Allocate C in device memory
	Matrix d_C;
	d_C.width = C.width; d_C.height = C.height;
	size = C.width * C.height * sizeof(float);
	hipMalloc(&d_C.elements, size);
	// Invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
	MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	// Read C from device memory
	hipMemcpy(C.elements, d_C.elements, size , hipMemcpyDeviceToHost);
	print(C);
	// Free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}
// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C){
	// Each thread computes one element of C
	// by accumulating results into Cvalue
	float Cvalue = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	for (int e = 0; e < A.width; ++e)
		Cvalue += A.elements[row * A.width + e] * B.elements[e * B.width + col];
	C.elements[row * C.width + col] = Cvalue;
}

int main( void ) {
   	Matrix A , B , C;
	A.width = B.width = C.width = N;
	A.height = B.height = C.height = N;
   	A.elements = (float *)malloc( N * N * sizeof(float) );
   	B.elements = (float *)malloc( N * N * sizeof(float) );
   	C.elements = (float *)malloc( N * N * sizeof(float) );
   	
   	for(int i = 0 ; i < N ;i++)
   		for(int j = 0 ; j < N ; j++)
   			A.elements[i*N + j] = (i==j) , B.elements[i*N + j] = (i==j);
   	
   	/*for(int i = 0 ; i < N ;i++)
   		for(int j = 0 ; j < N ; j++){
   			float r = 0;
   			for(int k = 0 ; k < N ; k++)
   				r += A.elements[i*N + k] * B.elements[k*N + j];
   				
   			C.elements[i*N + j] = r;
   		}
   	*/
   	MatMul( A , B , C );
}
