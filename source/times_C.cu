
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;
#define pi (2.0*acos(0.0))
#define eps 1e-6
#define ll long long
#define inf (1<<29)
#define vi vector<int>
#define vll vector<ll>
#define sc(x) scanf("%d",&x)
#define scl(x) scanf("%lld",&x)
#define all(v) v.begin() , v.end()
#define me(a,val) memset( a , val ,sizeof(a) )
#define pb(x) push_back(x)
#define pii pair<int,int> 
#define mp(a,b) make_pair(a,b)
#define Q(x) (x) * (x)
#define L(x) ((x<<1) + 1)
#define R(x) ((x<<1) + 2)
#define M(x,y) ((x+y)>>1)
#define fi first
#define se second
#define MOD 1000000007
#define ios ios::sync_with_stdio(0)
#define N 1024
#define TB 32


int main(){

	clock_t tStart = clock();

	int *a , *b , *c;
	size_t size = N * N * sizeof(int) ;
	a = (int *)malloc( size );
	b = (int *)malloc( size );
	c = (int *)malloc( size );
	
	for(int i = 0; i < N; i++)
		for(int j = 0; j < N; j++)
			a[i * N + j] = i + j;
			

	for(int i = 0; i < N; i++)
		for(int j = 0; j < N; j++)
			b[i * N + j] = 1;
	
	for(int i = 0; i < N; i++)
		for(int j = 0; j < N; j++)
			c[i * N + j] = a[i * N + j] * b[i * N + j];
	
	printf("Time taken: %.2fms\n", 1000.0 * (double)(clock() - tStart)/CLOCKS_PER_SEC);
	
	return 0;
	
}
