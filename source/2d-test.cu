#include "hip/hip_runtime.h"
#include<bits/stdc++.h>
using namespace std;

__global__ void add(int * dev_a[], int * dev_b[], int * dev_c[])
{
	dev_c[threadIdx.x][blockIdx.x]=dev_a[threadIdx.x][blockIdx.x]+dev_b[threadIdx.x][blockIdx.x];
     
}

__global__ void add2(int * dev_a, int * dev_b, int * dev_c)
{
	dev_c[threadIdx.x + blockDim.x * blockIdx.x]=dev_a[threadIdx.x + blockDim.x * blockIdx.x]+dev_b[threadIdx.x + blockDim.x * blockIdx.x];
     
}

inline void GPUassert(hipError_t code, char * file, int line, bool Abort=true)
{
    if (code != 0) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),file,line);
        if (Abort) exit(code);
    }       
}

#define GPUerrchk(ans) { GPUassert((ans), __FILE__, __LINE__); }

#define N 60

int aa[N][N];
int bb[N][N];
int cc[N][N];

int main(void)
{
    
    for(int i = 0; i < N; i++)
    	for(int j = 0; j < N; j++) aa[i][j] = 1, bb[i][j] = 2;
   
    
    int ts1 = clock();

    int ** h_a = (int **)malloc(N * sizeof(int *));
    for(int i=0; i<N;i++){
        GPUerrchk(hipMalloc((void**)&h_a[i], N*sizeof(int)));
        GPUerrchk(hipMemcpy(h_a[i], &aa[i][0], N*sizeof(int), hipMemcpyHostToDevice));
    }

    int **d_a;
    GPUerrchk(hipMalloc((void ***)&d_a, N * sizeof(int *)));
    GPUerrchk(hipMemcpy(d_a, h_a, N*sizeof(int *), hipMemcpyHostToDevice));

    int ** h_b = (int **)malloc(N * sizeof(int *));
    for(int i=0; i<N;i++){
        GPUerrchk(hipMalloc((void**)&h_b[i], N*sizeof(int)));
        GPUerrchk(hipMemcpy(h_b[i], &bb[i][0], N*sizeof(int), hipMemcpyHostToDevice));
    }

    int ** d_b;
    GPUerrchk(hipMalloc((void ***)&d_b, N * sizeof(int *)));
    GPUerrchk(hipMemcpy(d_b, h_b, N*sizeof(int *), hipMemcpyHostToDevice));

    int ** h_c = (int **)malloc(N * sizeof(int *));
    for(int i=0; i<N;i++){
        GPUerrchk(hipMalloc((void**)&h_c[i], N*sizeof(int)));
    }

    int ** d_c;
    GPUerrchk(hipMalloc((void ***)&d_c, N * sizeof(int *)));
    GPUerrchk(hipMemcpy(d_c, h_c, N*sizeof(int *), hipMemcpyHostToDevice));

	
    add<<<N,N>>>(d_a,d_b,d_c);
    int tf1 = clock();
    
    printf("time1: %.5lf\n", (tf1-ts1)/double(CLOCKS_PER_SEC)*1000);
    
    GPUerrchk(hipPeekAtLastError());

    for(int i=0; i<N;i++){
        GPUerrchk(hipMemcpy(&cc[i][0], h_c[i], N*sizeof(int), hipMemcpyDeviceToHost));
    }

    /*for(int i=0;i<N;i++) {
        for(int j=0;j<N;j++) {
            printf("(%d,%d):%d\n",i,j,cc[i][j]);
        }
    }*/
    
    int ts2 = clock();
    
    int *dev_a, *dev_b, *dev_c;
    
    hipMalloc((void **) &dev_a, N * N * sizeof(int));
    hipMalloc((void **) &dev_b, N * N * sizeof(int));
    hipMalloc((void **) &dev_c, N * N * sizeof(int));
    
    int *a = new int[N], *b = new int[N], *c = new int[N];
    
    for(int i = 0; i < N; i++)
    	for(int j = 0; j < N; j++) a[i * N + j] = aa[i][j], b[i * N + j] = bb[i][j];
    
    hipMemcpy(dev_a, a, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * N * sizeof(int), hipMemcpyHostToDevice);
    
	
    add2<<<N,N>>>(dev_a,dev_b,dev_c);
    int tf2 = clock();
    
    printf("time2: %.5lf\n", (tf2-ts2)/double(CLOCKS_PER_SEC)*1000);
        
    GPUerrchk(hipMemcpy(c, dev_c, N*N*sizeof(int), hipMemcpyDeviceToHost));

    /*for(int i=0;i<N;i++) {
        for(int j=0;j<N;j++) {
            printf("(%d,%d):%d\n",i,j,c[i * N + j]);
        }
    }*/

    return hipDeviceReset();
}
