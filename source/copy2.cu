
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;
#define pi (2.0*acos(0.0))
#define eps 1e-6
#define ll long long
#define inf (1<<29)
#define vi vector<int>
#define vll vector<ll>
#define sc(x) scanf("%d",&x)
#define scl(x) scanf("%lld",&x)
#define all(v) v.begin() , v.end()
#define me(a,val) memset( a , val ,sizeof(a) )
#define pb(x) push_back(x)
#define pii pair<int,int> 
#define mp(a,b) make_pair(a,b)
#define Q(x) (x) * (x)
#define L(x) ((x<<1) + 1)
#define R(x) ((x<<1) + 2)
#define M(x,y) ((x+y)>>1)
#define fi first
#define se second
#define MOD 1000000007
#define ios ios::sync_with_stdio(0)

typedef struct CudaMatrixStruct {
	double *data;
	int height, width;
} CudaMatrix;

__global__ void CudaPrintMatrix(CudaMatrix *A){
	
	printf("Elementos de CudaMatrix:\n");
	for(int i = 0; i < A->height * A->width ; i++)
		printf("%d: %.3lf\n", i, A->data[i]);
	
	
	printf("CudaMatrix->height: %d  CudaMatrix->width: %d\n", A->height, A->width);
}

void CudaCreateMatrix(CudaMatrix *&AA,int hei,int wid){	
	

	
	double *d;// = new double[hei*wid];
	d = (double *)malloc( hei * wid * sizeof(double) );
	for(int i = 0 ; i < hei * wid ; i++)
		d[i] = i + 1.0;
	
	double *data;
	hipMalloc( (void **) &data, sizeof(double) * hei * wid );
	hipMemcpy( data , d , sizeof(double) * hei * wid , hipMemcpyHostToDevice );
	
	CudaMatrix *A = new CudaMatrix();
	A->width = wid;
	A->height = hei;
	hipMalloc((void **)&AA, sizeof(CudaMatrix));
	
	hipMemcpy(AA, A, sizeof(CudaMatrix), hipMemcpyHostToDevice);
	hipMemcpy(&(AA->data), &data, sizeof(double *), hipMemcpyHostToDevice);
	
	
	CudaPrintMatrix<<<1, 1>>>(AA);
	
	//cudaFree(AA);
	//cudaFree(data);
}

int main() {
	CudaMatrix *devGr;
	int side = 1;
	CudaCreateMatrix( devGr , 2 * side + 1 , 2 * side + 1 );
	
	//CudaPrintMatrix<<<1,1>>>(devGr);//IMPRIME LA MATRICES CREADA
	
	hipFree(devGr);
}
