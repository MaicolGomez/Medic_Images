
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;
#define pi (2.0*acos(0.0))
#define eps 1e-6
#define ll long long
#define inf (1<<29)
#define vi vector<int>
#define vll vector<ll>
#define sc(x) scanf("%d",&x)
#define scl(x) scanf("%lld",&x)
#define all(v) v.begin() , v.end()
#define me(a,val) memset( a , val ,sizeof(a) )
#define pb(x) push_back(x)
#define pii pair<int,int> 
#define mp(a,b) make_pair(a,b)
#define Q(x) (x) * (x)
#define L(x) ((x<<1) + 1)
#define R(x) ((x<<1) + 2)
#define M(x,y) ((x+y)>>1)
#define fi first
#define se second
#define MOD 1000000007
#define ios ios::sync_with_stdio(0)
#define N 64
#define TB 32

float A[N][N];
float B[N][N];
float C[N][N];

__global__ void MatAdd(float A[N][N], float B[N][N], float C[N][N])
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;
if (i < N && j < N)
C[i][j] = A[i][j] + B[i][j];
}

int main(){

	float (*d_A)[N]; //pointers to arrays of dimension N
	float (*d_B)[N];
	float (*d_C)[N];

	for(int i = 0; i < N; i++) {
		for(int j = 0; j < N; j++) {
		    A[i][j] = i;
		    B[i][j] = j;
		}
	}       

	//allocation
	hipMalloc((void**)&d_A, (N*N)*sizeof(float));
	hipMalloc((void**)&d_B, (N*N)*sizeof(float));
	hipMalloc((void**)&d_C, (N*N)*sizeof(float));

	//copying from host to device
	hipMemcpy(d_A, A, (N*N)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, (N*N)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_C, C, (N*N)*sizeof(float), hipMemcpyHostToDevice);

	// Kernel invocation
	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);
	MatAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);

	//copying from device to host
	hipMemcpy(A, (d_A), (N*N)*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(B, (d_B), (N*N)*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(C, (d_C), (N*N)*sizeof(float), hipMemcpyDeviceToHost);

	for(int i = 0; i < N; i++){
		for(int j = 0; j < N; j++)
			printf("%lf ", C[i][j]);
		
		printf("\n");
	}
	return 0;
}
